#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(fabs, y[index] = fabs(x[index]));

__global__ void popc_kernel(const int n, const float* a,
    const float* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popc(static_cast<uint32_t>(a[index]) ^
                      static_cast<uint32_t>(b[index]));
  }
}

__global__ void popcll_kernel(const int n, const double* a,
    const double* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popcll(static_cast<uint64_t>(a[index]) ^
                      static_cast<uint64_t>(b[index]));
  }
}

template <>
uint32_t caffe_gpu_hamming_distance<float>(const int n, const float* x,
                                  const float* y) {
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popc_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
uint32_t caffe_gpu_hamming_distance<double>(const int n, const double* x,
                                   const double* y) {
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popcll_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        /* NOLINT_NEXT_LINE(build/include_what_you_use) */
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

}  // namespace caffe
