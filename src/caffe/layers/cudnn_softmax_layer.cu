
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  cudnnStatus_t stat = cudnnSoftmaxForward(handle_, CUDNN_SOFTMAX_ACCURATE,
      CUDNN_SOFTMAX_MODE_CHANNEL,
      bottom_desc_, bottom_data, top_desc_, top_data);
  CHECK_EQ(stat,CUDNN_STATUS_SUCCESS) << "Error in cudnnSoftmaxBackward.";
}

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = (*bottom)[0]->gpu_data();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    cudnnStatus_t stat = cudnnSoftmaxBackward(handle_, CUDNN_SOFTMAX_ACCURATE,
        CUDNN_SOFTMAX_MODE_CHANNEL,
        top_desc_, top_data, top_desc_, top_diff, bottom_desc_, bottom_diff));
    CHECK_EQ(stat,CUDNN_STATUS_SUCCESS) << "Error in cudnnSoftmaxBackward.";
  }
}

INSTANTIATE_CLASS(CuDNNSoftmaxLayer);

}  // namespace caffe
#endif
