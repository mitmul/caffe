
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNSigmoidLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  cudnnStatus_t stat = cudnnActivationForward(this->handle_,
      CUDNN_ACTIVATION_SIGMOID,
      this->bottom_desc_, bottom_data, this->top_desc_, top_data);
  CHECK_EQ(stat,CUDNN_STATUS_SUCCESS)
      << "Error in cudnnActivationForward.";
}

template <typename Dtype>
void CuDNNSigmoidLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = (*bottom)[0]->gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  cudnnStatus_t stat = cudnnActivationBackward(this->handle_,
      CUDNN_ACTIVATION_SIGMOID,
      this->top_desc_, top_data, this->top_desc_, top_diff,
      this->bottom_desc_, bottom_data, this->bottom_desc_, bottom_diff);
  CHECK_EQ(stat,CUDNN_STATUS_SUCCESS)
      << "Error in cudnnActivationBackward.";
}

INSTANTIATE_CLASS(CuDNNSigmoidLayer);

}  // namespace caffe
#endif
